#include "hip/hip_runtime.h"
﻿
#include <sstream>
#include <iomanip>
#include <stdio.h>
#include <cmath>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "Kernel.cuh"

__device__ const uint32_t& Max( const uint32_t& a, const uint32_t& b )
{
  return a >= b ? a : b;
}

__device__ uint8_t Component( const uint32_t& color, const uint32_t& idx )
{
  switch ( idx )
  {
    case 0:
      return static_cast<uint8_t>( ( color & 0x000000FF ) >> 0 );
    case 1:
      return static_cast<uint8_t>( ( color & 0x0000FF00 ) >> 8 );
    case 2:
      return static_cast<uint8_t>( ( color & 0x00FF0000 ) >> 16 );
    case 3:
      return static_cast<uint8_t>( ( color & 0xFF000000 ) >> 24 );
    default:
      return 0;
  }
}

__device__ uint32_t Color( const uint8_t r = 0, const uint8_t g = 0, const uint8_t b = 0, const uint8_t a = 255 )
{
  return ( r << 0 ) | ( g << 8 ) | ( b << 16 ) | ( a << 24 );
}

__device__ uint32_t GetValue( uint32_t* buffer, int32_t x, int32_t y, uint32_t width, uint32_t height )
{
  const int32_t rx = x < 0 ? width - 1 : x >= width ? 0 : x;
  const int32_t ry = y < 0 ? height - 1 : y >= height ? 0 : y;
  return buffer[rx + ry * width]; // RGBA
}

__global__ void StepKernel( uint32_t* frontBuffer, uint32_t* backBuffer, const uint32_t width, const uint32_t height, const uint32_t livingColor, const uint32_t deadColor )
{
  static const bool mDecideData[] = {
  //0  1  2  3  4  5  6  7  8    living neighbour count
    0, 0, 0, 1, 0, 0, 0, 0, 0,   // dead cell new state |  8
    0, 0, 1, 1, 0, 0, 0, 0, 0 }; // live cell new state | 12

  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

  if ( x >= width || y >= height )
  {
    return;
  }

  const uint32_t current = ( GetValue( frontBuffer, x, y, width, height ) == deadColor ? 0 : 1 );
  const uint32_t sum = ( GetValue( frontBuffer, x - 1, y - 1, width, height ) == deadColor ? 0 : 1 ) +
                       ( GetValue( frontBuffer, x    , y - 1, width, height ) == deadColor ? 0 : 1 ) +
                       ( GetValue( frontBuffer, x + 1, y - 1, width, height ) == deadColor ? 0 : 1 ) +
                       ( GetValue( frontBuffer, x - 1, y    , width, height ) == deadColor ? 0 : 1 ) +
                       ( GetValue( frontBuffer, x + 1, y    , width, height ) == deadColor ? 0 : 1 ) +
                       ( GetValue( frontBuffer, x - 1, y + 1, width, height ) == deadColor ? 0 : 1 ) +
                       ( GetValue( frontBuffer, x    , y + 1, width, height ) == deadColor ? 0 : 1 ) +
                       ( GetValue( frontBuffer, x + 1, y + 1, width, height ) == deadColor ? 0 : 1 );
  backBuffer[x + width * y] = ( mDecideData[current * 9 + sum] ? livingColor : deadColor );
}

__global__ void FillKernel( uint32_t* buffer, const uint32_t width, const uint32_t height, const uint32_t value )
{
  const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if ( x >= width || y >= height )
  {
    return;
  }
  const size_t offset = x + width * y;
  buffer[offset] = value;
}

__global__ void InitRandom( unsigned int seed, const uint32_t width, const uint32_t height, hiprandState_t* states )
{
  const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if ( x >= width || y >= height )
  {
    return;
  }

  /* we have to initialize the state */
  hiprand_init( seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
               x, /* the sequence number should be different for each core (unless you want all
                           cores to get the same sequence of numbers for some reason - use thread id! */
               y, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
               &states[x + y * width] );
}

__global__ void RandomKernel( uint32_t* buffer, const uint32_t width, const uint32_t height, const uint32_t livingColor, const uint32_t deadColor, const float prob, hiprandState_t* states )
{
  const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if ( x >= width || y >= height )
  {
    return;
  }

  const float r = hiprand_uniform( &states[x + y * width] );
  uint32_t v = deadColor;
  if ( r > prob )
  {
    v = livingColor;
  }

  buffer[x + width * y + 0] = Max(v, buffer[x + width * y + 0]);
}

hipError_t RunFillKernel( uint32_t* buffer, const uint32_t value, const uint32_t width, const uint32_t height )
{
  const dim3 blockSize( 32, 32 );// number of threads per block along x/y-axis
  const dim3 gridSize( ( width + blockSize.x - 1 ) / blockSize.x
                       , ( height + blockSize.y - 1 ) / blockSize.y ); // number of blocks in the grid

  //hipEvent_t start, stop;
  //hipEventCreate( &start );
  //hipEventCreate( &stop );
  //
  //hipEventRecord( start, 0 );
  FillKernel<<<gridSize, blockSize>>>( buffer, width, height, value );
  //hipEventRecord( stop, 0 );
  //hipEventSynchronize( stop );
  //
  //float time = 0.0f;
  //hipEventElapsedTime( &time, start, stop );

  return hipGetLastError();
}

hipError_t RunStepKernel( uint32_t* frontBuffer, uint32_t* backBuffer, uint32_t width, uint32_t height, const uint32_t livingColor, const uint32_t deadColor )
{
  const dim3 blockSize( 32, 32 );// number of threads per block along x/y-axis
  const dim3 gridSize( (width + blockSize.x - 1) / blockSize.x
                       , (height + blockSize.y - 1) / blockSize.y ); // number of blocks in the grid

  //hipEvent_t start, stop;
  //hipEventCreate( &start );
  //hipEventCreate( &stop );
  //
  //hipEventRecord( start, 0 );
  StepKernel<<<gridSize, blockSize>>>( frontBuffer, backBuffer, width, height, livingColor, deadColor );
  //hipEventRecord( stop, 0 );
  //hipEventSynchronize( stop );
  //
  //float time = 0.0f;
  //hipEventElapsedTime( &time, start, stop );
  return hipGetLastError();
}

hipError_t RunRandomKernel( uint32_t* buffer, const float prob, const uint32_t width, const uint32_t height, const uint32_t livingColor, const uint32_t deadColor )
{
  const dim3 blockSize( 32, 32 );// number of threads per block along x/y-axis
  const dim3 gridSize( (width + blockSize.x - 1) / blockSize.x
                       , (height + blockSize.y - 1) / blockSize.y ); // number of blocks in the grid

  // TODO: revisit this random number stuff. Random states per pixel?
  hiprandState_t* states = nullptr;
  hipMalloc((void**) &states, width * height * sizeof(hiprandState_t));

  InitRandom<<<gridSize, blockSize>>>( static_cast<uint32_t>( time( nullptr ) ), width, height, states );
  RandomKernel<<<gridSize, blockSize>>>( buffer, width, height, livingColor, deadColor, prob, states );

  hipFree( states );

  return hipGetLastError();
}
